
#include <hip/hip_runtime.h>
#include <stdio.h>

// CUDA kernel to print a greeting message
__global__ void cuda_greetings() {
    printf("Where are the greetings\n");
}

int main() {
    // Launch the kernel with 1 block and 1 thread
    cuda_greetings<<<1, 1>>>();
    
    // Ensure the kernel is executed before program exit
    hipDeviceSynchronize();

    // Check for CUDA errors
    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(error));
        return 1;
    }

    return 0;
}



